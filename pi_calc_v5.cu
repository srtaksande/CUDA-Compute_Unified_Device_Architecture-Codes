
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<sys/time.h>

#define N 99999999
#define NUM_THDS 512

__global__ void calc_area(double dx, double *aread)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	double x,y;
	if(i<N)
	{	
		x = i*dx;
		y = sqrt(1-x*x);
		aread[i] = y*dx;
	}
}
__global__ void calc_reduction(double *aread, double *aread2)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int working_thds;
	__shared__ double tmp_area[NUM_THDS];
	tmp_area[threadIdx.x] = 0.0;
	if(i<N)
	{	
		tmp_area[threadIdx.x] = aread[i];
	}
	__syncthreads();
	if(i<N)
	{	
		working_thds = NUM_THDS;
		while(1)
		{
			working_thds = working_thds/2;
			if(threadIdx.x < working_thds)
			{
				tmp_area[threadIdx.x] = tmp_area[threadIdx.x] + tmp_area[threadIdx.x + working_thds]; 
			}
			if(working_thds == 1)
			{
				break;
			}
			__syncthreads();
		}
		if(threadIdx.x == 0)
		{		
			aread2[blockIdx.x] = tmp_area[0];
		}
	}

}
int main()
{
	int i;
	double total_area, pi, *area, *aread,*aread2;
	double dx;
	double exe_time;
	struct timeval stop_time, start_time;
	
	dx = 1.0/N;
	total_area = 0.0;
	
	gettimeofday(&start_time, NULL);
	
	int num_threads_per_block = NUM_THDS;
	int total_threads = N;
	int num_blocks = total_threads / num_threads_per_block + 1;
	
	area = (double *)malloc(num_blocks*sizeof(double));
	hipMalloc(&aread, N*sizeof(double));
	hipMalloc(&aread2, num_blocks*sizeof(double));
	
	calc_area<<<num_blocks,num_threads_per_block>>>(dx, aread);
	calc_reduction<<<num_blocks,num_threads_per_block>>>(aread,aread2);

	
	hipMemcpy(area,aread2,num_blocks*sizeof(double),hipMemcpyDeviceToHost);
	
	for(i=0;i<num_blocks;i++)
	{
		total_area += area[i];	
	}
	/*for(i=0;i<N;i++)
	{
		x = i*dx;
		y = sqrt(1-x*x);
		area += y*dx;
	}*/
	
	gettimeofday(&stop_time, NULL);	
	exe_time = (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
	
	pi = 4.0*total_area;
	printf("\n Value of pi is = %.16lf\n Execution time is = %lf seconds\n", pi, exe_time);
	
	free(area);
	hipFree(aread);
	
}

