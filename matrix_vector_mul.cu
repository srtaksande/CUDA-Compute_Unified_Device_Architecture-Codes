
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

#define VECTORSIZE 5000

__global__ void matrix_vec_mul(int *A, int *B, int *C)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j;
	int sum = 0;
	if(i<VECTORSIZE)
	{
		for(j=0;j<VECTORSIZE;j++)
		{
			sum = sum + A[i*VECTORSIZE+j]*B[i];	
		}
		C[i] =  sum;
	}
}


int main(int argc, char **argv)
{
	int myid, size;
	int i, j, sum;
	int *A, *B, *C, *Ad, *Bd, *Cd; 	
	double exe_time;
	struct timeval stop_time, start_time;
	
	//Allocate and initialize the arrays
	A = (int *)malloc(VECTORSIZE*VECTORSIZE*sizeof(int));
	B = (int *)malloc(VECTORSIZE*sizeof(int));
	C = (int *)malloc(VECTORSIZE*sizeof(int));
	
	//Initialize data to some value
	for(i=0;i<VECTORSIZE;i++)
	{
		for(j=0;j<VECTORSIZE;j++)
		{
			A[i*VECTORSIZE+j] = 1;	
		}
		B[i] = 1;
	}
	
	//print the data
	/*printf("\nInitial data: \n");
	for(i=0;i<VECTORSIZE;i++)
	{
		for(j=0;j<VECTORSIZE;j++)
		{
			printf("\t%d ", A[i*VECTORSIZE+j]);	
		}
		printf("\n");
	}
	printf("\n");
	for(i=0;i<VECTORSIZE;i++)
	{
		printf("\t%d", B[i]);
	}*/	
	
	gettimeofday(&start_time, NULL);
	
	/*for(i=0;i<VECTORSIZE;i++)
	{
		sum = 0;
		for(j=0;j<VECTORSIZE;j++)
		{
			sum = sum + A[i*VECTORSIZE+j]*B[i];	
		}
		C[i] =  sum;
	}*/
	
	hipMalloc(&Ad,VECTORSIZE*VECTORSIZE*sizeof(int));
	hipMalloc(&Bd,VECTORSIZE*sizeof(int));
	hipMalloc(&Cd,VECTORSIZE*sizeof(int));
	
	hipMemcpy(Ad,A,VECTORSIZE*VECTORSIZE*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(Bd,B,VECTORSIZE*sizeof(int),hipMemcpyHostToDevice);
	
	int Total_num_Threads = VECTORSIZE;
	int num_threads_per_block = 5;
	int numblocks = Total_num_Threads/num_threads_per_block + 1;
	
	matrix_vec_mul<<<numblocks,num_threads_per_block>>>(Ad,Bd,Cd);
	
	hipMemcpy(C,Cd,VECTORSIZE*sizeof(int),hipMemcpyDeviceToHost);
	
	gettimeofday(&stop_time, NULL);	
	exe_time = (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
	
	//print the data
	/*printf("\n Matrix-Vector Multiplication output: \n");
	for(i=0;i<VECTORSIZE;i++)
	{
		printf("\t%d", C[i]);	
	}*/
	printf("\n Execution time is = %lf seconds\n", exe_time);
	
	printf("\nProgram exit!\n");
	
	
	hipFree(Ad);
	hipFree(Bd);
	hipFree(Cd);
	
	//Free arrays
	free(A); 
	free(B);
	free(C);
}
