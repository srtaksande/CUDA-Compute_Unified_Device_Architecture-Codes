
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>


__global__ void arradd(int* md, int* nd, int* pd, int size)
{
	__shared__ int ms[400], ns[400];
	int myid = threadIdx.x;
	
	ms[myid] = md[myid];
	ns[myid] = nd[myid];

	__syncthreads();

	pd[myid] = ms[myid] + ns[myid];
}


int main()
{
	int size = 400 * sizeof(int);
	int m[400], n[400], p[400],*md, *nd,*pd;
	int i=0;

	
	for(i=0; i<400; i++ )
	{
		m[i] = i;
		n[i] = i;
		p[i] = 0;
	}

	hipMalloc(&md, size);
	hipMemcpy(md, m, size, hipMemcpyHostToDevice);

	hipMalloc(&nd, size);
	hipMemcpy(nd, n, size, hipMemcpyHostToDevice);

	hipMalloc(&pd, size);

	dim3   DimGrid(1, 1);     
	dim3   DimBlock(400, 1);   


	arradd<<< DimGrid,DimBlock >>>(md,nd,pd,size);

	hipMemcpy(p, pd, size, hipMemcpyDeviceToHost);
	hipFree(md); 
	hipFree(nd);
	hipFree (pd);

	for(i=0; i<400; i++ )
	{
		printf("\t%d",p[i]);
	}	

}




