
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__device__ int calc_square(int val)
{
	int val_square;
	val_square = val*val;
	return val_square;
}


__global__ void calc(int *aa, int *a, int problem_size)
{
	int i = blockIdx.x*blockDim.x+ threadIdx.x;
	if(i < problem_size)
	{
		aa[i] = calc_square(a[i]);
	}
}


int main()
{
	int size = 400 * sizeof(int);
	int a[400], aa[400], *ad, *aad;
	int i=0;
	

	//Initialize the vectors
	for(i=0; i<400; i++ )
	{
		a[i] = i;
	}

	// Add two vectors
	//for(i=0; i<400; i++ )
	//{
	//	aa[i] = calc_square(a[i]);		
	//}
	
	hipMalloc(&ad,size);
	hipMalloc(&aad,size);	
	
	int total_threads = 400;
	int num_threads_per_block = 256;
	int num_blocks = total_threads / num_threads_per_block + 1;
	
	hipMemcpy(ad,a,size,hipMemcpyHostToDevice);
	
	calc<<<num_blocks,num_threads_per_block>>>(aad,ad,400);
	
	hipMemcpy(aa,aad,size,hipMemcpyDeviceToHost);
	
	// print the output
	for(i=0; i<400; i++ )
	{
		printf("\t%d",aa[i]);
	}	
}




