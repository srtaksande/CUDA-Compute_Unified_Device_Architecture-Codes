
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void arradd(int* md, int* nd, int* pd)
{
	int myid = threadIdx.y * blockDim.x + threadIdx.x;
	
	pd[myid] = md[myid] + nd[myid];
}


int main()
{
	int size = 20 *20* sizeof(int);
	int m[20][20], n[20][20], p[20][20],*md, *nd,*pd;
	int i=0,j=0;

	
	for(i=0; i<20; i++ )
	{
		for(j=0; j<20; j++ )
		{
			m[i][j] = i;
			n[i][j] = i;
			p[i][j] = 0;
		}
	}

	hipMalloc(&md, size);
	hipMemcpy(md, m, size, hipMemcpyHostToDevice);

	hipMalloc(&nd, size);
	hipMemcpy(nd, n, size, hipMemcpyHostToDevice);

	hipMalloc(&pd, size);

	dim3   DimGrid(1, 1);     
	dim3   DimBlock(20, 20);   


	arradd<<< DimGrid,DimBlock >>>(md,nd,pd);

	hipMemcpy(p, pd, size, hipMemcpyDeviceToHost);
	hipFree(md); 
	hipFree(nd);
	hipFree (pd);

	for(i=0; i<20; i++ )
	{
		for(j=0; j<20; j++ )
		{
			printf("\t%d",p[i][j]);
		}
		printf("\n");
	}
		
	

}




