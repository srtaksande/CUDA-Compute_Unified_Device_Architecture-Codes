
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<sys/time.h>

#define N 1000000
/*
                N  PRIME_NUMBER

                1           0
               10           4
              100          25
            1,000         168
           10,000       1,229
          100,000       9,592
        1,000,000      78,498
       10,000,000     664,579
      100,000,000   5,761,455
    1,000,000,000  50,847,534

*/

__global__ void prime_count(int *countd)
{
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	int flag = 0;
	int j;
	if((i>2)&&(i<N))
	{
		for(j=2;j<i;j++)	
	    	{
		    if((i%j) == 0)
		    {
			    flag = 1;
			    break;
		    }
	    	}
        	if(flag == 0)
        	{
            		countd[i] = 1;
        	}
        	else
        	{
            		countd[i] = 0;
        	}
        }
}



int main()
{
	int i, j;
	int count, flag, *cnt, *cntd;
	double exe_time;
	struct timeval stop_time, start_time;
	
	count = 1; // 2 is prime. Our loop starts from 3
	
	gettimeofday(&start_time, NULL);
	
	/*for(i=3;i<N;i++)
	{
	 	flag = 0;
		for(j=2;j<i;j++)	
	    	{
		    if((i%j) == 0)
		    {
			    flag = 1;
			    break;
		    }
	    	}
        	if(flag == 0)
        	{
            		count++;
        	}
	}*/
	cnt = (int *)malloc(N*sizeof(int));
	hipMalloc(&cntd,N*sizeof(int));
	
	int num_threads_per_block = 256;
	int total_threads = N;
	int num_blocks = total_threads / num_threads_per_block + 1;  
	
	prime_count<<<num_blocks,num_threads_per_block>>>(cntd);
	
	hipMemcpy(cnt,cntd,N*sizeof(int),hipMemcpyDeviceToHost);
	
	for(i=3;i<N;i++)
	{
		if(cnt[i]==1)
		{
			count++;
		}
	}
	
	
	gettimeofday(&stop_time, NULL);	
	exe_time = (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
	
	printf("\n Number of prime numbers = %d \n Execution time is = %lf seconds\n", count, exe_time);
	
	free(cnt);
	hipFree(cntd);
	
}

