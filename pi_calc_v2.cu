
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<sys/time.h>

#define N 99999999
#define NUM_THDS 128

__global__ void calc_area(double dx, double *aread)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	double x,y;
	double tmp;
	__shared__ double tmp_area[NUM_THDS];
	tmp_area[threadIdx.x] = 0.0;
	if(i<N)
	{	
		x = i*dx;
		y = sqrt(1-x*x);
		tmp_area[threadIdx.x] = y*dx;
	}
	__syncthreads();
	if(i<N)
	{
		if(threadIdx.x == 0)
		{
			tmp = 0.0;
			for(int j=0;j<NUM_THDS;j++)
			{
				tmp += tmp_area[j];
			}
			aread[blockIdx.x] = tmp;
		}
	}
}

int main()
{
	int i;
	double total_area, pi, *area, *aread;
	double dx;
	double exe_time;
	struct timeval stop_time, start_time;
	
	dx = 1.0/N;
	total_area = 0.0;
	
	gettimeofday(&start_time, NULL);
	
	int num_threads_per_block = NUM_THDS;
	int total_threads = N;
	int num_blocks = total_threads / num_threads_per_block + 1;
	
	area = (double *)malloc(num_blocks*sizeof(double));
	hipMalloc(&aread, num_blocks*sizeof(double));
	
	calc_area<<<num_blocks,num_threads_per_block>>>(dx, aread);
	
	hipMemcpy(area,aread,num_blocks*sizeof(double),hipMemcpyDeviceToHost);
	
	for(i=0;i<num_blocks;i++)
	{
		total_area += area[i];	
	}
	/*for(i=0;i<N;i++)
	{
		x = i*dx;
		y = sqrt(1-x*x);
		area += y*dx;
	}*/
	
	gettimeofday(&stop_time, NULL);	
	exe_time = (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
	
	pi = 4.0*total_area;
	printf("\n Value of pi is = %.16lf\n Execution time is = %lf seconds\n", pi, exe_time);
	
	free(area);
	hipFree(aread);
	
}

