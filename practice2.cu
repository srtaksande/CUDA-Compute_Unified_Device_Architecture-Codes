
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void calc(double *c, double *a, double *b, double alpha)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	c[i] = a[i] + alpha*b[i];	
}

int main()
{
	int size = 400 * sizeof(double);
	double a[400], b[400], c[400], *ad, *bd, *cd, alpha;
	int i=0;
	
	alpha = 0.001;

	for(i=0; i<400; i++ )
	{
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}

	hipMalloc(&ad, size);
	hipMalloc(&bd, size);
	hipMalloc(&cd, size);
	
	hipMemcpy(ad,a,size,hipMemcpyHostToDevice);
	hipMemcpy(bd,b,size,hipMemcpyHostToDevice);
	
	//for(i=0; i<400; i++ )
	//{
	//	c[i] = a[i] + alpha*b[i];		
	//}
	
	dim3 blocks(10,1);
	dim3 threads(40,1);
	
	calc<<<blocks,threads>>>(cd,ad,bd,alpha);
	
	hipMemcpy(c,cd,size,hipMemcpyDeviceToHost);

	for(i=0; i<400; i++ )
	{
		printf("\t%lf",c[i]);
	}	
}




