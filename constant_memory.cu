
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__constant__ int md[200], nd[200];

__global__ void arradd(int* pd, int size)
{
	int myid = threadIdx.x;
	
	pd[myid] = md[myid] + nd[myid];
}

int main()
{
	int size = 200 * sizeof(int);
	int m[200], n[200], p[200],*pd;
	int i=0;

	
	for(i=0; i<200; i++ )
	{
		m[i] = i;
		n[i] = i;
		p[i] = 0;
	}

	hipMemcpyToSymbol(HIP_SYMBOL(md), m, size);
	hipMemcpyToSymbol(HIP_SYMBOL(nd), n, size);

	hipMalloc(&pd, size);

	dim3   DimGrid(1, 1);     
	dim3   DimBlock(200, 1);   


	arradd<<< DimGrid,DimBlock >>>(pd,size);

	hipMemcpy(p, pd, size, hipMemcpyDeviceToHost);

	hipFree(pd);

	for(i=0; i<200; i++ )
	{
		printf("\t%d",p[i]);
	}	

}




