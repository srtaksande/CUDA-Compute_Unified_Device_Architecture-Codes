
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#define N 200

__global__ void arradd(int* md, int* nd, int* pd)
{
	int myid = threadIdx.x;
	
	pd[myid] = md[myid] + nd[myid];
}


int main()
{
	int *m, *n, *p;
	int i=0;

        hipMallocManaged(&m, N*sizeof(int));
	hipMallocManaged(&n, N*sizeof(int));
	hipMallocManaged(&p, N*sizeof(int));
	
	for(i=0; i<N; i++ )
	{
		m[i] = i;
		n[i] = i;
		p[i] = 0;
	}  
	arradd<<< 1,N >>>(m,n,p);
	hipDeviceSynchronize();
	for(i=0; i<N; i++ )
	{
		printf("\t%d",p[i]);
	}	
	hipFree(m);
	hipFree(n);
	hipFree(p);	
}

