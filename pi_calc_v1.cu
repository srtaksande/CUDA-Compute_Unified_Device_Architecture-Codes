
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<sys/time.h>

#define N 99999999

__global__ void calc_area(double dx, double *aread)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	double x,y;
	if(i<N)
	{	
		x = i*dx;
		y = sqrt(1-x*x);
		aread[i] = y*dx;
	}
}

int main()
{
	int i;
	double total_area, pi, *area, *aread;
	double dx;
	double exe_time;
	struct timeval stop_time, start_time;
	
	dx = 1.0/N;
	total_area = 0.0;
	
	gettimeofday(&start_time, NULL);
	
	int num_threads_per_block = 256;
	int total_threads = N;
	int num_blocks = total_threads / num_threads_per_block + 1;
	
	area = (double *)malloc(N*sizeof(double));
	hipMalloc(&aread, N*sizeof(double));
	
	calc_area<<<num_blocks,num_threads_per_block>>>(dx, aread);
	
	hipMemcpy(area,aread,N*sizeof(double),hipMemcpyDeviceToHost);
	
	for(i=0;i<N;i++)
	{
		total_area += area[i];	
	}
	/*for(i=0;i<N;i++)
	{
		x = i*dx;
		y = sqrt(1-x*x);
		area += y*dx;
	}*/
	
	gettimeofday(&stop_time, NULL);	
	exe_time = (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
	
	pi = 4.0*total_area;
	printf("\n Value of pi is = %.16lf\n Execution time is = %lf seconds\n", pi, exe_time);
	
	free(area);
	hipFree(aread);
	
}

